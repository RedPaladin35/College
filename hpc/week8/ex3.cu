
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

typedef struct {
    int start;
    int length;
} WordInfo;

__device__ bool compareWords(const char* sen, int start, int l, const char* target, int tl){
    if(l!=tl) return false;
    for(int i=0; i<l; i++){
        if(sen[start+i]!=target[i]) return false;
    }
    return true;
}

__global__ void countWords(const char* sen, int l, WordInfo* words, int wc, const char* target, int tl, int* d_count){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < wc){
        if(compareWords(sen, words[idx].start, words[idx].length, target, tl)){
            atomicAdd(d_count, 1);
        }
    }
}

int main(){
    char h_str[] = "what when what when what";
    char h_target[] = "what";

    int l = strlen(h_str);
    int tl = strlen(h_target);

    int i=0, wc=0;
    int maxWords = 100;

    WordInfo* h_words = (WordInfo*)malloc(maxWords * sizeof(WordInfo));

    while(i<l){
        while(i<l && h_str[i]==' ') i++;
        if(i>=l) break;

        int start = i;
        while(i<l && h_str[i]!=' ') i++;
        int length = i - start;

        h_words[wc].start = start;
        h_words[wc].length = length;
        wc++;

        if(wc > maxWords) break;
    }

    int h_count, zero=0;

    char *d_str, *d_target;
    WordInfo* d_words;
    int *d_count;

    hipMalloc((void**)&d_str, l*sizeof(char));
    hipMalloc((void**)&d_target, tl*sizeof(char));
    hipMalloc((void**)&d_words, wc*sizeof(WordInfo));
    hipMalloc((void**)&d_count, sizeof(int));

    hipMemcpy(d_str, h_str, l*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_target, h_target, tl*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_words, h_words, wc*sizeof(WordInfo), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &zero, sizeof(int), hipMemcpyHostToDevice);

    countWords<<<1, 256>>>(d_str, l, d_words, wc, d_target, tl, d_count);
    hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    printf("The word %s appears %d times.\n", h_target, h_count);

    hipFree(d_str);
    hipFree(d_target);
    hipFree(d_words);
    hipFree(d_count);
    free(h_words);
}