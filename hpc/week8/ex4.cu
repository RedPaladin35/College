
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

__global__ void reverseString(char* d_str, int l){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int half = l/2;

    if(idx < half){
        char temp = d_str[idx];
        d_str[idx] = d_str[l-idx-1];
        d_str[l-idx-1] = temp;
    }
}

int main(){
    char h_str[] = "I am a third year student";
    int l = strlen(h_str);

    char* d_str;
    hipMalloc((void**)&d_str, l*sizeof(char));
    hipMemcpy(d_str, h_str, l*sizeof(char), hipMemcpyHostToDevice);

    reverseString<<<1, 256>>> (d_str, l);
    hipMemcpy(h_str, d_str, l*sizeof(char), hipMemcpyDeviceToHost);
    printf("%s\n", h_str);
    hipFree(d_str);
}