
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<string.h>
#include<stdlib.h>

__global__ void repeatString(char* d_output, char* d_input, int l, int N){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<N){
        int sp = idx * l;
        for(int i=0; i<l; i++){
            d_output[sp+i] = d_input[i];
        }
    }
}

int main(){
    const char* s = "Hello";
    int N = 3;

    int l = strlen(s);
    int ol = l*N;

    char h_output[ol+1];
    h_output[ol] = '\0';

    char *d_input, *d_output;
    hipMalloc((void**)&d_input, l*sizeof(char));
    hipMalloc((void**)&d_output, ol*sizeof(char));

    hipMemcpy(d_input, s, l*sizeof(char), hipMemcpyHostToDevice);

    repeatString<<<1, 256>>>(d_output, d_input, l, N);
    hipMemcpy(h_output, d_output, ol*sizeof(char), hipMemcpyDeviceToHost);

    printf("%s\n", h_output);
    hipFree(d_input);
    hipFree(d_output);
}