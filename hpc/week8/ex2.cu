
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

typedef struct {
    int start;
    int length;
} WordInfo;

__global__ void reverseWords(char* d_str, WordInfo* d_words, int N){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < N){
        int start = d_words[idx].start;
        int length = d_words[idx].length;

        for(int i=0; i<length/2; i++){
            char temp = d_str[start+i];
            d_str[start+i] = d_str[start+length-1-i];
            d_str[start+length-i-1] = temp;
        }
    }
}

int main(){
    char h_str[] = "I am a third year student";
    int l = strlen(h_str);
    printf("Original string:\n");
    printf("%s\n", h_str);
    
    int maxWords = 100;
    WordInfo* words = (WordInfo*)malloc(maxWords*sizeof(WordInfo));

    int wc=0;
    int i=0;

    while(i<l){
        while(i<l && h_str[i]==' ') i++;
        if(i>l) break;
        
        int start = i;
        while(i<l && h_str[i]!=' ') i++;
        int length = i-start;

        words[wc].start = start;
        words[wc].length = length;
        wc++;

        if(wc>maxWords) break;
    }

    char* d_str;
    WordInfo* d_words;
    hipMalloc((void**)&d_str, l*sizeof(char));
    hipMalloc((void**)&d_words, wc*sizeof(WordInfo));

    hipMemcpy(d_str, h_str, l*sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_words, words, wc*sizeof(WordInfo), hipMemcpyHostToDevice);

    reverseWords<<<1, 256>>>(d_str, d_words, wc);
    hipMemcpy(h_str, d_str, l*sizeof(char), hipMemcpyDeviceToHost);
    printf("After reversing:\n");
    printf("%s\n", h_str);

    hipFree(d_str);
    hipFree(d_words);
    free(words);
}