
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void transformMatrix(float* arr, int m, int n){
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int id = r*n + c;
    if(r<m && c<n){
        int p = r+1;
        int val = arr[id];
        int res = 1.0f;
        for(int i=0; i<p; i++){
            res *= val;
        }
        arr[id] = res;
    }
}

int main(){
    int m, n;
    scanf("%d", &m);
    scanf("%d", &n);
    float h_matrix[m*n];
    for(int i=0; i<m*n; i++){
        scanf("%f", &h_matrix[i]);
    }
    int s = m*n*sizeof(float);
    float *d_matrix;
    hipMalloc((void**)&d_matrix, s);
    hipMemcpy(d_matrix, h_matrix, s, hipMemcpyHostToDevice);
    dim3 blockDim(m, n, 1);
    transformMatrix<<<1, blockDim>>>(d_matrix, m, n);
    hipMemcpy(h_matrix, d_matrix, s, hipMemcpyDeviceToHost);
    for(int i=1; i<=m*n; i++){
        printf("%f\t", h_matrix[i-1]);
        if(i%3 == 0){
            printf("\n");
        }
    }
    hipFree(d_matrix);
}