#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void rowMul(float* d_a, float* d_b, float* d_res, int m, int n, int k){
    int r = blockIdx.x * blockDim.x + threadIdx.x;
    if(r<m){
        for(int c=0; c<n; c++){
            float sum = 0.0f;
            for(int x=0; x<k; x++){
                sum += d_a[r*k+x] * d_b[x*n+c]; 
            }
            d_res[r*n+c] = sum;
        }
    }
}

__global__ void colMul(float* d_a, float* d_b, float* d_res, int m, int n, int k){
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if(c<n){
        for(int r=0; r<m; r++){
            float sum = 0.0f;
            for(int x=0; x<k; x++){
                sum += d_a[r*k+x] * d_b[x*n+c];
            }
            d_res[r*n+c] = sum;
        }
    }
}

__global__ void elemMul(float* d_a, float* d_b, float* d_res, int m, int n, int k){
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if(r<m && c<n){
        float sum = 0.0f;
        for(int x=0; x<k; x++){
            sum += d_a[r*k+x] * d_b[x*n+c];
        }
        d_res[r*n+c] = sum;
    }
}

int main(){
    int m=3, n=3;
    float a[m][n] = {{1, 2, 3}, {1, 2, 3}, {1, 2, 3}};
    float b[m][n] = {{1, 2, 2}, {1, 2, 3}, {1, 2, 3}};
    float res[m][n];
    int s = m*n*sizeof(float);

    float *d_a, *d_b, *d_res;
    hipMalloc((void**)&d_a, s);
    hipMalloc((void**)&d_b, s);
    hipMalloc((void**)&d_res, s);

    hipMemcpy(d_a, a, s, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, s, hipMemcpyHostToDevice);

    rowMul<<<1, 3>>>(d_a, d_b, d_res, m, n, 3);
    hipMemcpy(res, d_res, s, hipMemcpyDeviceToHost);
    for(int i=0; i<m; i++){
        for(int j=0; j<n; j++){
            printf("%f\t", res[i][j]);
        }
        printf("\n");
    }
    printf("\n");

    colMul<<<1, 3>>>(d_a, d_b, d_res, m, n, 3);
    hipMemcpy(res, d_res, s, hipMemcpyDeviceToHost);
    for(int i=0; i<m; i++){
        for(int j=0; j<n; j++){
            printf("%f\t", res[i][j]);
        }
        printf("\n");
    }
    printf("\n");

    dim3 blockDim(m, n, 1);
    elemMul<<<1, blockDim>>>(d_a, d_b, d_res, m, n, 3);
    hipMemcpy(res, d_res, s, hipMemcpyDeviceToHost);
    for(int i=0; i<m; i++){
        for(int j=0; j<n; j++){
            printf("%f\t", res[i][j]);
        }
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);
}