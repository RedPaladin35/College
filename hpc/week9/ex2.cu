
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void addRows(float* a, float* b, float* res, int m, int n){
    int r = blockDim.x * blockIdx.x + threadIdx.x;
    if(r<m){
        for(int c=0; c<n; c++){
            res[r*n+c] = a[r*n+c] + b[r*n+c];
        }
    }
}

__global__ void addCols(float* a, float* b, float* res, int m, int n){
    int c = blockDim.x * blockIdx.x + threadIdx.x;
    if(c<n){
        for(int r=0; r<m; r++){
            res[r*n+c] = a[r*n+c] + b[r*n+c];
        }
    }
}

__global__ void addElems(float* a, float* b, float* res, int m, int n){
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    if(r<m && c<n){
        res[r*n+c] = a[r*n+c] + b[r*n+c];
    }
}

int main(){
    int m = 3;
    int n = 3;
    float a[9] = {1, 2, 3, 1, 2, 3, 1, 2, 3};
    float b[9] = {1, 2, 3, 1, 2, 3, 1, 2, 3};
    float res[9];
    int s = m*n*sizeof(float);

    float *d_a, *d_b, *d_res;
    hipMalloc((void**)&d_a, s);
    hipMalloc((void**)&d_b, s);
    hipMalloc((void**)&d_res, s);

    hipMemcpy(d_a, a, s, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, s, hipMemcpyHostToDevice);

    addRows<<<1, 3>>>(d_a, d_b, d_res, m, n);
    hipMemcpy(res, d_res, s, hipMemcpyDeviceToHost);
    for(int i=1; i<=m*n; i++){
        printf("%f\t", res[i-1]);
        if(i%3==0){
            printf("\n");
        }
    }
    printf("\n");

    addCols<<<1, 3>>>(d_a, d_b, d_res, m, n);
    hipMemcpy(res, d_res, s, hipMemcpyDeviceToHost);
    for(int i=1; i<=m*n; i++){
        printf("%f\t", res[i-1]);
        if(i%3==0){
            printf("\n");
        }
    }
    printf("\n");

    dim3 blockDim(m, n, 1);
    addElems<<<1, blockDim>>>(d_a, d_b, d_res, m, n);
    hipMemcpy(res, d_res, s, hipMemcpyDeviceToHost);
    for(int i=1; i<=m*n; i++){
        printf("%f\t", res[i-1]);
        if(i%3==0){
            printf("\n");
        }
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);
}