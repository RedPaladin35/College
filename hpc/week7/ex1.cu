
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 10

__global__ void add(int* a, int* b, int* c){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N){
        c[id] = a[id] + b[id];
    }
}

void printVector(int* a){
    for(int i=0; i<N; i++){
        printf("%d\t", a[i]);
    }
    printf("\n");
}

int main(){
    size_t s = N * sizeof(int);

    int a[N], b[N], c[N];

    for(int i=0; i<N; i++){
        a[i] = i*2 + 3;
        b[i] = i*3 + 2;
    }

    printf("A values:\n");
    printVector(a);

    printf("B values:\n");
    printVector(b);

    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, s);
    hipMalloc((void**)&d_b, s);
    hipMalloc((void**)&d_c, s);

    hipMemcpy(d_a, a, s, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, s, hipMemcpyHostToDevice);

    printf("Value of N is %d\n", N);
    printf("Running with 1 block and N threads:\n");
    add<<<1, N>>>(d_a, d_b, d_c);
    hipMemcpy(c, d_c, s, hipMemcpyDeviceToHost);
    printVector(c);
    printf("--------\n");

    printf("Running with N blocks each having 1 thread:\n");
    add<<<N, 1>>>(d_a, d_b, d_c);
    hipMemcpy(c, d_c, s, hipMemcpyDeviceToHost);
    printVector(c);
    printf("--------\n");

    int tpb = 256;
    int bpg = (N + tpb - 1) / tpb;
    printf("Running with %d blocks and %d threads:\n", bpg, tpb);
    add<<<bpg, tpb>>>(d_a, d_b, d_c);
    hipMemcpy(c, d_c, s, hipMemcpyDeviceToHost);
    printVector(c);
    printf("--------\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}