
#include <hip/hip_runtime.h>
#include<stdio.h>

#define N 16
#define M 5

__global__ void conv(const float* input, const float* mask, float* output){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i<N){
        float res = 0.0f;
        for(int j=0; j<M; j++){
            int idx = i - j;
            if(idx >= 0){
                res += input[idx] * mask[j];
            }
        }
        output[i] = res;
    }
}

void printVector(float* c){
    for(int i=0; i<N; i++){
        printf("%f\t", c[i]);
    }
    printf("\n");
}

int main(){
    size_t is = N * sizeof(float);
    size_t ms = M * sizeof(float);

    float h_input[N], h_mask[M], h_output[N];
    for(int i=0; i<N; i++){
        h_input[i] = i + 1;
    }
    for(int i=0; i<M; i++){
        h_mask[i] = 0.2f;
    }

    float *d_input, *d_mask, *d_output;
    hipMalloc((void**)&d_input, is);
    hipMalloc((void**)&d_mask, ms);
    hipMalloc((void**)&d_output, is);

    hipMemcpy(d_input, h_input, is, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, h_mask, ms, hipMemcpyHostToDevice);

    int tpb = 256;
    int bpg = (N + tpb - 1) / tpb;
    conv<<<bpg, tpb>>>(d_input, d_mask, d_output);
    hipMemcpy(h_output, d_output, is, hipMemcpyDeviceToHost);

    printf("Input:\n");
    printVector(h_input);

    printf("Mask:\n");
    for(int i=0; i<5; i++){
        printf("%f\t", h_mask[i]);
    }
    printf("\n");

    printf("Output:\n");
    printVector(h_output);

    hipFree(d_input);
    hipFree(d_mask);
    hipFree(d_output);
}