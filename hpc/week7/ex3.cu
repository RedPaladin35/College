
#include <hip/hip_runtime.h>
#include<stdio.h>

#define N 10 

__global__ void computeSine(const float* input, float* output){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N){
        output[id] = sinf(input[id]);
    }
}

void printVector(float* a){
    for(int i=0; i<N; i++){
        printf("%f\t", a[i]);
    }
    printf("\n");
}

int main(){
    float h_input[N], h_output[N];
    size_t s = N * sizeof(float);

    for(int i=0; i<N; i++){
        h_input[i] = i * 0.1f;
    }

    float *d_input, *d_output;
    hipMalloc((void**)&d_input, s);
    hipMalloc((void**)&d_output, s);

    hipMemcpy(d_input, h_input, s, hipMemcpyHostToDevice);
    
    computeSine<<<1, N>>>(d_input, d_output);
    hipMemcpy(h_output, d_output, s, hipMemcpyDeviceToHost);

    printf("Input in radians:\n");
    printVector(h_input);

    printf("Output in sine:\n");
    printVector(h_output);

    hipFree(d_input);
    hipFree(d_output);
}